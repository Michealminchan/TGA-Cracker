

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

__device__ void mystrcpy(char dest[], const char source[]) 
{
    int i = 0;
    while ((dest[i] = source[i]) != '\0')
    {
        i++;
    } 
}
__device__ int mystrcmp(char string1[], char string2[] )
{
    for (int i = 0; ; i++)
    {
        if (string1[i] != string2[i])
        {
            return string1[i] < string2[i] ? -1 : 1;
        }
        if (string1[i] == '\0')
        {
            return 0;
        }
    }
}

__device__ size_t mystrlen(const char *str)
{
        const char *s;
        for (s = str; *s; ++s);
        return (s - str);
}

__device__ int dResult = 0;

__device__ void permuteKRec(char *alphabet, char *prefix, int n, int k, char *pwd)
{
	if(dResult == 1) return;
    if (k == 0) 
    {
        //printf("%s\n", prefix);
        if(mystrcmp(prefix, pwd) == 0)
            dResult = 1;
		return;
    }
    for (int i = 0; i < n; ++i)
    {
        size_t len = mystrlen(prefix);
        char newPrefix[100]; // = malloc(len + 1 + 1 );
        mystrcpy(newPrefix, prefix);
        newPrefix[len] = alphabet[i];
        newPrefix[len + 1] = '\0';   
    
        permuteKRec(alphabet, newPrefix, n, k - 1, pwd);
        free( newPrefix );
    }
}


__global__ void permuteK(char *dAlphabet, char *dPermut, int k, char *pwd){
    

    if(dResult == 1) return;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 1)
	printf("dento de permute");
    int pos = i*3;
    char prefix[4];
    prefix[0] = dPermut[pos];
    prefix[1] = dPermut[pos+1];
    prefix[2] = dPermut[pos+2];
    prefix[3] = '\0';
    permuteKRec(dAlphabet, prefix, 26, k-3, pwd);

    //if(result == 1) dResult = 1;
}


void permute3(char *permutation)
{
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    int n = 26;
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < n; ++j){
            for (int k = 0; k < n; ++k){
                char newPrefix[4];
                
                newPrefix[0] = alphabet[i];
                newPrefix[1] = alphabet[j];
                newPrefix[2] = alphabet[k];
                newPrefix[3] = '\0';              
                strcat(permutation, newPrefix);
            }
        }
    }
}

int permute1and2(char *pwd){
	return 0;
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char** argv){
    
    //Buscar la palabra dentro de un diccionario, words.txt
    
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    fp = fopen("words.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    int result = 0;
    int cont = 0;
    while ((read = getline(&line, &len, fp)) != -1 && result == 0) {
        line[read-1] = '\0';
        ++cont;
        if(strcmp(line, argv[1]) == 0)
            result = 1;
    }
    fclose(fp);
    if (line)
        free(line);
   
    
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    char hPermut[26*26*26*4] = "";
    
    //Fuerza bruta
    int k = 1;
    unsigned long int numBytes;
    char *dPermut;
    char *dAlphabet; 
    char *dPwd;
    int nThreads = 1024;
    unsigned long int nBlocks;


    while(result == 0 && k < 10){
        printf("Probando con palabras de longitud %d\n" , k);
        if(k < 3)
            result = permute1and2(argv[1]);
	if(k == 3){
		permute3(hPermut);
            numBytes = 26*26*26*4*sizeof(char);
            hipMalloc((char**)&dPermut, numBytes);
            hipMalloc((char**)&dAlphabet, 26*sizeof(char));
	    hipMalloc((char**)&dPwd, 26*sizeof(char));
            //cudaMalloc((int*)&dResult, sizeof(int)); 
            CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 
		
		printf("CudaMallocs Done\n");
            // Copiar datos desde el host en el device 
            hipMemcpy(dPermut, hPermut, numBytes, hipMemcpyHostToDevice);
            hipMemcpy(dAlphabet, alphabet, 26*sizeof(char), hipMemcpyHostToDevice);
	    hipMemcpy(dPwd, argv[1], 26*sizeof(char), hipMemcpyHostToDevice);
  
            //cudaMemcpy(dResult, 0, sizeof(int), cudaMemcpyHostToDevice);
            printf("CudaMemcpy done\n");
            CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
        }
        if(k >= 3){
			unsigned long int N = pow(26, 3);
			nBlocks = (N + nThreads - 1)/nThreads;
		    dim3 dimGrid(nBlocks, 1, 1);
			dim3 dimBlock(nThreads, 1, 1);
		printf("call func\n");
            permuteK<<<dimGrid, dimBlock>>>(dAlphabet, dPermut, k, dPwd);
            printf("ret func\n");
		CheckCudaError((char *) "Invocar Kernel", __LINE__);

			hipMemcpyFromSymbol(&result, HIP_SYMBOL("dResult"), sizeof(result), 0, hipMemcpyDeviceToHost);
	    //cudaMemcpy(result, dResult, sizeof(int), cudaMemcpyDeviceToHost);
	    //CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

        }
            
        
        ++k;
    }
    //result = 1 si la encuentra
    printf("%d\n", result);
}
