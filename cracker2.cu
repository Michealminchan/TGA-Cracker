#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>



__global__ void permuteK(char *dAlphabet, char *dPermut, int k, int dResult, char *pwd){
    
    __shared__ int result;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i/3;
    char prefix[4];
    prefix[0] = dPermut[pos];
    prefix[1] = dPermut[pos+1];
    prefix[2] = dPermut[pos+2];
    prefix[3] = '\0';
    result = printAllKLengthRec(alphabet, prefix, 26, k-3, pwd);
}

int printAllKLengthRec(char *alphabet, char *prefix, int n, int k, char *pwd)
{
    int result = 0;
    int size = (int)sizeof(prefix)/sizeof(prefix[0]);
    // Base case: k is 0,
    if (k == 0) 
    {
        //printf("%s\n", prefix);
        if(strcmp(prefix, pwd) == 0)
            result = 1;
        return result;
    }
 
    // One by one add all characters 
    // from alphabet and recursively 
    // call for k equals to k-1
    for (int i = 0; i < n; ++i)
    {
        size_t len = strlen(prefix);
        char *newPrefix = malloc(len + 1 + 1 );
        strcpy(newPrefix, prefix);
        newPrefix[len] = alphabet[i];
        newPrefix[len + 1] = '\0';
        
    
        int r = printAllKLengthRec(alphabet, newPrefix, n, k - 1, pwd);
        free( newPrefix );
        if(r == 1)
            return 1;
    }
    return 0;
}


void permute3(char *permutation)
{
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    int n = 26;
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < n; ++j){
            for (int k = 0; k < n; ++k){
                char newPrefix[4];
                
                newPrefix[0] = alphabet[i];
                newPrefix[1] = alphabet[j];
                newPrefix[2] = alphabet[k];
                newPrefix[3] = '\0';              
                strcat(permutation, newPrefix);
            }
        }
    }
}

/*
void permute(char *permut3){
    permut3[0] = 'a';
    permut3[1] = '\0';
    permut3[2] = 'b';
    permut3[3] = 'c';
    permut3[4] = '\0';
}*/


int main(int argc, char** argv){
    
    //Buscar la palabra dentro de un diccionario, words.txt
    
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    fp = fopen("words.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    int result = 0;
    int cont = 0;
    while ((read = getline(&line, &len, fp)) != -1 && result == 0) {
        line[read-1] = '\0';
        ++cont;
        if(strcmp(line, argv[1]) == 0)
            result = 1;
    }
    fclose(fp);
    if (line)
        free(line);
   
    
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    char hPermut[26*26*26*4] = "";
    
    //Fuerza bruta
    int k = 1;
    float numBytes;
    char *dPermut;
    char *dAlphabet;
    while(result == 0 && k < 10){
        printf("Probando con palabras de longitud %d\n" , k);
        if(k < 3)
            result = printAllKLengthRec(alphabet, "", 26, k, argv[1]);
        if(k == 3){
            permute3(hPermut);
            numBytes = 26*26*26*3*sizeof(char)
            hipMalloc((char**)&dPermut, numBytes);
            hipMalloc((char**)&dAlphabet, 26*sizeof(char));
            hipMalloc((int*)&dResult, sizeof(int)); 
            CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 

            // Copiar datos desde el host en el device 
            hipMemcpy(dPermut, hPermut, numBytes, hipMemcpyHostToDevice);
            hipMemcpy(dAlphabet, alphabet, 26*sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(dResult, 0, sizeof(int), hipMemcpyHostToDevice);
            
            CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
        }
        if(k >= 3){
            //cuda......
            permuteK<<<dimGrid, dimBlock>>>(dAlphabet, dPermut, k, dResult);
            CheckCudaError((char *) "Invocar Kernel", __LINE__);
        }
            
        
        ++k;
    }
    //result = 1 si la encuentra
    printf("%d\n", result);
}
