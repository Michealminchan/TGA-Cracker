
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

__device__ void mystrcpy(char dest[], const char source[])
{
    int i = 0;
    while ((dest[i] = source[i]) != '\0')
    {
        i++;
    }
}
__device__ int mystrcmp(char string1[], char string2[] )
{
    for (int i = 0; ; i++)
    {
        if (string1[i] != string2[i])
        {
            return string1[i] < string2[i] ? -1 : 1;
        }
        if (string1[i] == '\0')
        {
            return 0;
        }
    }
}

__device__ size_t mystrlen(const char *str)
{
        const char *s;
        for (s = str; *s; ++s);
        return (s - str);
}

__device__ int dResult = 0;
__device__ char dPass[30] = "";

__device__ void permuteKRec(char *alphabet, char *prefix, int n, int k, char *pwd)
{
        if(dResult == 1) return;
    if (k == 0)
    {
        //printf("%s\n", prefix);
        if(mystrcmp(prefix, pwd) == 0){
            dResult = 1;
            mystrcpy(dPass, prefix);
        }
        return;
    }
    for (int i = 0; i < n; ++i)
    {
        size_t len = mystrlen(prefix);
        char newPrefix[100]; // = malloc(len + 1 + 1 );
        mystrcpy(newPrefix, prefix);
        newPrefix[len] = alphabet[i];
        newPrefix[len + 1] = '\0';

        permuteKRec(alphabet, newPrefix, n, k - 1, pwd);
        free( newPrefix );
    }
}


__global__ void permuteK(char *dAlphabet, char *dPermut, int k, char *pwd){


    if(dResult == 1) return;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
  
    int pos = i*3;
    char prefix[4];
    prefix[0] = dPermut[pos];
    prefix[1] = dPermut[pos+1];
    prefix[2] = dPermut[pos+2];
    prefix[3] = '\0';
    permuteKRec(dAlphabet, prefix, 26, k-3, pwd);

    //if(result == 1) dResult = 1;
}


void permute3(char *permutation)
{
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    int n = 26;
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < n; ++j){
            for (int k = 0; k < n; ++k){
                char newPrefix[4];

                newPrefix[0] = alphabet[i];
                newPrefix[1] = alphabet[j];
                newPrefix[2] = alphabet[k];
                newPrefix[3] = '\0';
                strcat(permutation, newPrefix);
            }
        }
    }
}

int permute1and2(char *pwd, char *guessed, int k){
	char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    int n = 26;
	char word[3];
    for (int i = 0; i < n; ++i){
		word[0] = alphabet[i];
		if(k == 1){
			word[1] = '\0';
		    if(strcmp(word, pwd) == 0){
		        strcpy(guessed, word);
				return 1;
		    }
		}
		else{
		    for (int j = 0; j < n; ++j){
				word[1] = alphabet[j];
				word[2] = '\0';
				if(strcmp(word, pwd) == 0){
				    strcpy(guessed, word);
					return 1;
		   		}
			}
		}
	}
	return 0;
}

void CheckCudaError(char sms[], int line) {
  hipError_t error;

  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char** argv){

    //Buscar la palabra dentro de un diccionario, words.txt
    printf("empezamos\n");
    char passwordGuessed[15];
    /*FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    fp = fopen("words.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    int result = 0;
    int cont = 0;
    while ((read = getline(&line, &len, fp)) != -1 && result == 0) {
        line[read-1] = '\0';
        ++cont;
        if(strcmp(line, argv[1]) == 0){
            result = 1;
			strcpy(passwordGuessed, argv[1]);
		}
    }
    fclose(fp);
    if (line)
        free(line);*/
	int result = 0;

    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    char hPermut[26*26*26*4] = "";

    //Fuerza bruta
    int k = 1;
    unsigned long int numBytes;
    char *dPermut;
    char *dAlphabet;
    char *dPwd;
    int nThreads = 1024;
    unsigned long int nBlocks;

	hipEvent_t E0, E1;
	float TiempoTotal;

	hipEventCreate(&E0);
 	hipEventCreate(&E1);

	hipEventRecord(E0, 0);
 	hipEventSynchronize(E0);

    while(result == 0 && k < 15){
        printf("Probando con palabras de longitud %d\n" , k);
        if(k < 3)
            result = permute1and2(argv[1], passwordGuessed, k);
        if(k == 3){
            permute3(hPermut);
            numBytes = 26*26*26*4*sizeof(char);
            hipMalloc((char**)&dPermut, numBytes);
            hipMalloc((char**)&dAlphabet, 26*sizeof(char));
            hipMalloc((char**)&dPwd, 26*sizeof(char));
            //cudaMalloc((int*)&dResult, sizeof(int));
            CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);

            // Copiar datos desde el host en el device
            hipMemcpy(dPermut, hPermut, numBytes, hipMemcpyHostToDevice);
            hipMemcpy(dAlphabet, alphabet, 26*sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(dPwd, argv[1], 26*sizeof(char), hipMemcpyHostToDevice);

            //cudaMemcpy(dResult, 0, sizeof(int), cudaMemcpyHostToDevice);
            CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
        }
        if(k >= 3){
            unsigned long int N = pow(26, 3);
            nBlocks = (N + nThreads - 1)/nThreads;
            dim3 dimGrid(nBlocks, 1, 1);
            dim3 dimBlock(nThreads, 1, 1);
            permuteK<<<dimGrid, dimBlock>>>(dAlphabet, dPermut, k, dPwd);
	    printf("permutao\n");
            CheckCudaError((char *) "Invocar Kernel", __LINE__);

            hipMemcpyFromSymbol(&result, HIP_SYMBOL(dResult), sizeof(int), 0, hipMemcpyDeviceToHost);
            if(result == 1){
               hipMemcpyFromSymbol(&passwordGuessed, HIP_SYMBOL(dPass), k*sizeof(char), 0, hipMemcpyDeviceToHost);           
            }

        }
        ++k;
    }
	

	
	hipEventRecord(E1, 0);
 	hipEventSynchronize(E1);	
	hipEventElapsedTime(&TiempoTotal,  E0, E1);
	hipEventDestroy(E0); hipEventDestroy(E1);	

	if(k >= 3){
		// Liberar Memoria del device 
  		hipFree(dPermut); hipFree(dAlphabet); hipFree(dPwd);
		hipDeviceSynchronize();
	}

	if(result == 1){
		printf("Tiempo paralelo para encontrar la contraseña %s: %4.6f segundos\n", passwordGuessed, TiempoTotal/1000.0f);	
	}
	else
		printf("Password not found...\n");
    
}
