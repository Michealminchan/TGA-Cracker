#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

int printAllKLengthRec(char *alphabet, char *prefix, int n, int k, char *pwd)
{
    int result = 0;
    //int size = (int)sizeof(prefix)/sizeof(prefix[0]);
    // Base case: k is 0,
    if (k == 0) 
    {
        //printf("%s\n", prefix);
        if(strcmp(prefix, pwd) == 0)
            result = 1;
        return result;
    }
 
    // One by one add all characters 
    // from alphabet and recursively 
    // call for k equals to k-1
    for (int i = 0; i < n; ++i)
    {
        size_t len = strlen(prefix);
        char newPrefix[30]= "";// = malloc(len + 1 + 1 );
        strcpy(newPrefix, prefix);
        newPrefix[len] = alphabet[i];
        newPrefix[len + 1] = '\0';
        
    
        int r = printAllKLengthRec(alphabet, newPrefix, n, k - 1, pwd);
        free( newPrefix );
        if(r == 1)
            return 1;
    }
    return 0;
}

__global__ void permuteK(char *dAlphabet, char *dPermut, int k, int &dResult, char *pwd){
    
    if(dResult == 1) return;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i*3;
    char prefix[4];
    prefix[0] = dPermut[pos];
    prefix[1] = dPermut[pos+1];
    prefix[2] = dPermut[pos+2];
    prefix[3] = '\0';
    int result = printAllKLengthRec(dAlphabet, prefix, 26, k-3, pwd);

    if(result == 1) dResult = 1;
}


void permute3(char *permutation)
{
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    int n = 26;
    for (int i = 0; i < n; ++i){
        for (int j = 0; j < n; ++j){
            for (int k = 0; k < n; ++k){
                char newPrefix[4];
                
                newPrefix[0] = alphabet[i];
                newPrefix[1] = alphabet[j];
                newPrefix[2] = alphabet[k];
                newPrefix[3] = '\0';              
                strcat(permutation, newPrefix);
            }
        }
    }
}

/*
void permute(char *permut3){
    permut3[0] = 'a';
    permut3[1] = '\0';
    permut3[2] = 'b';
    permut3[3] = 'c';
    permut3[4] = '\0';
}*/


void CheckCudaError(char sms[], int line) {
  hipError_t error;
 
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
}

int main(int argc, char** argv){
    
    //Buscar la palabra dentro de un diccionario, words.txt
    
    FILE * fp;
    char * line = NULL;
    size_t len = 0;
    ssize_t read;
    fp = fopen("words.txt", "r");
    if (fp == NULL)
        exit(EXIT_FAILURE);
    int result = 0;
    int cont = 0;
    while ((read = getline(&line, &len, fp)) != -1 && result == 0) {
        line[read-1] = '\0';
        ++cont;
        if(strcmp(line, argv[1]) == 0)
            result = 1;
    }
    fclose(fp);
    if (line)
        free(line);
   
    
    char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    char hPermut[26*26*26*4] = "";
    
    //Fuerza bruta
    int k = 1;
    unsigned long int numBytes;
    char *dPermut;
    char *dAlphabet; 
    char *dPwd;
    int dResult;
    int nThreads = 1024;
    unsigned long int nBlocks;


    while(result == 0 && k < 10){
        printf("Probando con palabras de longitud %d\n" , k);
        if(k < 3)
            result = printAllKLengthRec(alphabet, "", 26, k, argv[1]);
        if(k == 3){
            permute3(hPermut);
            numBytes = 26*26*26*3*sizeof(char);
            hipMalloc((char**)&dPermut, numBytes);
            hipMalloc((char**)&dAlphabet, 26*sizeof(char));
	    hipMalloc((char**)&dPwd, 26*sizeof(char));
            //hipMalloc((int*)&dResult, sizeof(int)); 
            CheckCudaError((char *) "Obtener Memoria en el device", __LINE__); 

            // Copiar datos desde el host en el device 
            hipMemcpy(dPermut, hPermut, numBytes, hipMemcpyHostToDevice);
            hipMemcpy(dAlphabet, alphabet, 26*sizeof(char), hipMemcpyHostToDevice);
	    hipMemcpy(dPwd, argv[1], 26*sizeof(char), hipMemcpyHostToDevice);
            //hipMemcpy(dResult, 0, sizeof(int), hipMemcpyHostToDevice);
            
            CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);
        }
        if(k >= 3){
	    unsigned long int N = pow(26, 3);
	    nBlocks = (N + nThreads - 1)/nThreads;
            dim3 dimGrid(nBlocks, nBlocks, 1);
	    dim3 dimBlock(nThreads, nThreads, 1);
            permuteK<<<dimGrid, dimBlock>>>(dAlphabet, dPermut, k, result, dPwd);
            CheckCudaError((char *) "Invocar Kernel", __LINE__);

	    //hipMemcpy(result, dResult, sizeof(int), hipMemcpyDeviceToHost);
	    //CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

        }
            
        
        ++k;
    }
    //result = 1 si la encuentra
    printf("%d\n", result);
}
